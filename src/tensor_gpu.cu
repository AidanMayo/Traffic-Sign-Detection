#include "hip/hip_runtime.h"
#include "../include/tensor.hpp"
#include "../include/logger.hpp"
#include <hip/hip_runtime.h>
#include <iostream>

__global__ void makeContiguousKernel(float*a, float*b, int* shape, int* strides, int dims, int totalSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx >= totalSize) return;

    int lin = idx;
    int aIdx = 0;

    for (int d = dims - 1; d >= 0; --d) {
        int cur = lin % shape[d];
        lin /= shape[d];
        aIdx += cur * strides[d];
    }

    b[idx] = a[aIdx];
}

void Tensor::makeContiguousGpu() {
    if (contiguous) return;

    float* newData;
    hipError_t err;

    err = hipMalloc(&newData, sizeof(float) * totalSize);
    if (err != hipSuccess) {
        LOG_CUDA_OP("MALLOC", "makeContiguous", 0, 0, false, hipGetErrorString(err));
        return;
    }
    LOG_MEMORY_ALLOC("GPU", totalSize * sizeof(float), "makeContiguous newData");

    int* dShape;
    int* dStrides;
    err = hipMalloc(&dShape, shape.size() * sizeof(int));
    if (err != hipSuccess) {
        LOG_CUDA_OP("MALLOC", "makeContiguous", 0, 0, false, hipGetErrorString(err));
        hipFree(newData);
        return;
    }
    
    err = hipMalloc(&dStrides, strides.size() * sizeof(int));
    if (err != hipSuccess) {
        LOG_CUDA_OP("MALLOC", "makeContiguous", 0, 0, false, hipGetErrorString(err));
        hipFree(newData);
        hipFree(dShape);
        return;
    }
    
    hipMemcpy(dShape, shape.data(), sizeof(int) * shape.size(), hipMemcpyHostToDevice);
    hipMemcpy(dStrides, strides.data(), sizeof(int) * strides.size(), hipMemcpyHostToDevice);

    int blockSize = 256;
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    makeContiguousKernel<<<gridSize, blockSize>>>(gpuData, newData, dShape, dStrides, shape.size(), totalSize);
    
    err = hipGetLastError();
    if (err != hipSuccess) {
        LOG_CUDA_OP("KERNEL", "makeContiguousKernel", blockSize, gridSize, false, hipGetErrorString(err));
    } else {
        LOG_CUDA_OP("KERNEL", "makeContiguousKernel", blockSize, gridSize, true, "");
    }

    hipFree(dShape);
    hipFree(dStrides);
    LOG_MEMORY_DEALLOC("GPU", (shape.size() + strides.size()) * sizeof(int), "makeContiguous temp arrays");

    hipFree(gpuData);
    LOG_MEMORY_DEALLOC("GPU", totalSize * sizeof(float), "makeContiguous old data");
    
    gpuData = newData;

    computeStrides();
    contiguous = true;
}


__global__ void fillKernel(float* data, int size, float val) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = val;
    }
}

void Tensor::fillGpu(float val) {
    int blockSize = 256;
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    fillKernel<<<gridSize, blockSize>>>(gpuData, totalSize, val);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        LOG_CUDA_OP("KERNEL", "fillKernel", blockSize, gridSize, false, hipGetErrorString(err));
        return;
    }
    
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        LOG_CUDA_OP("SYNC", "fillKernel", blockSize, gridSize, false, hipGetErrorString(err));
    } else {
        LOG_CUDA_OP("KERNEL", "fillKernel", blockSize, gridSize, true, "");
    }
}

__global__ void addTensorKernel(float*a, const float* b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] += b[idx];
}

void Tensor::addTensorGpu(const Tensor& other) {
    addTensorKernel<<<(totalSize + 255)/256, 256>>>(gpuData, other.gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void addScalarKernel(float*a, const float val, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] += val;
}

void Tensor::addScalarGpu(const float val) {
    addScalarKernel<<<(totalSize + 255)/256, 256>>>(gpuData, val, totalSize);
    hipDeviceSynchronize();
}

__global__ void addBiasKernel(float* a, const float* bias, int channels, int size, int totalSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalSize) {
        int c = (idx / size) % channels;
        a[idx] += bias[c];
    }
}

void Tensor::addBiasGpu(const Tensor& bias) {
    int blockSize = 256;
    int gridSize = (totalSize + blockSize - 1) / blockSize;
    addBiasKernel<<<gridSize, blockSize>>>(gpuData, bias.gpuData, shape[1], shape[2] * shape[3], totalSize);
    
    hipError_t err = hipGetLastError();
    if (err != hipSuccess) {
        LOG_CUDA_OP("KERNEL", "addBiasKernel", blockSize, gridSize, false, hipGetErrorString(err));
        return;
    }
    
    hipDeviceSynchronize();
    err = hipGetLastError();
    if (err != hipSuccess) {
        LOG_CUDA_OP("SYNC", "addBiasKernel", blockSize, gridSize, false, hipGetErrorString(err));
    } else {
        LOG_CUDA_OP("KERNEL", "addBiasKernel", blockSize, gridSize, true, "");
    }
}

__global__ void subtractTensorKernel(float*a, const float* b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] -= b[idx];
}

void Tensor::subtractTensorGpu(const Tensor& other) {
    subtractTensorKernel<<<(totalSize + 255)/256, 256>>>(gpuData, other.gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void subtractScalarKernel(float*a, const float val, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] -= val;
}

void Tensor::subtractScalarGpu(const float val) {
    subtractScalarKernel<<<(totalSize + 255)/256, 256>>>(gpuData, val, totalSize);
    hipDeviceSynchronize();
}

__global__ void multiplyTensorKernel(float*a, const float* b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] *= b[idx];
}

void Tensor::multiplyTensorGpu(const Tensor& other) {
    multiplyTensorKernel<<<(totalSize + 255)/256, 256>>>(gpuData, other.gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void multiplyScalarKernel(float*a, const float val, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] *= val;
}

void Tensor::multiplyScalarGpu(const float val) {
    multiplyScalarKernel<<<(totalSize + 255)/256, 256>>>(gpuData, val, totalSize);
    hipDeviceSynchronize();
}

__global__ void multiplyBiasKernel(float* a, const float* bias, int channels, int size, int totalSize) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < totalSize) {
        int c = (idx / size) % channels;
        a[idx] *= bias[c];
    }
}

void Tensor::multiplyBiasGpu(const Tensor& bias) {
    multiplyBiasKernel<<<(totalSize + 255)/256, 256>>>(gpuData, bias.gpuData, shape[1], shape[2] * shape[3], totalSize);
    hipDeviceSynchronize();
}

__global__ void divideTensorKernel(float*a, const float* b, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] /= b[idx];
}

void Tensor::divideTensorGpu(const Tensor& other) {
    divideTensorKernel<<<(totalSize + 255)/256, 256>>>(gpuData, other.gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void divideScalarKernel(float*a, const float val, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] /= val;
}

void Tensor::divideScalarGpu(const float val) {
    divideScalarKernel<<<(totalSize + 255)/256, 256>>>(gpuData, val, totalSize);
    hipDeviceSynchronize();
}

__global__ void negateKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = -a[idx];
}

void Tensor::negateGpu() {
    negateKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void ReLUKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = fmaxf(a[idx], 0);
}

void Tensor::ReLUGpu() {
    ReLUKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void sigmoidKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = 1.0 / (1.0 + expf(-1.0 * a[idx]));
}

void Tensor::sigmoidGpu() {
    sigmoidKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void tanhKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = tanhf(a[idx]);
}

void Tensor::tanhGpu() {
    tanhKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void LReLUKernel(float*a, float alpha, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = a[idx] < 0 ? alpha * a[idx] : a[idx];
}

void Tensor::LReLUGpu(float alpha) {
    LReLUKernel<<<(totalSize + 255)/256, 256>>>(gpuData, alpha, totalSize);
    hipDeviceSynchronize();
}

__global__ void ELUKernel(float*a, float alpha, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = a[idx] < 0 ? alpha * (expf(a[idx]) - 1) : a[idx];
}

void Tensor::ELUGpu(float alpha) {
    ELUKernel<<<(totalSize + 255)/256, 256>>>(gpuData, alpha, totalSize);
    hipDeviceSynchronize();
}

__global__ void squareKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = a[idx] * a[idx];
}

void Tensor::squareGpu() {
    squareKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void sqrtKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = sqrtf(a[idx]);
}

void Tensor::sqrtGpu() {
    sqrtKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void expKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = expf(a[idx]);
}

void Tensor::expGpu() {
    expKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void logKernel(float*a, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) a[idx] = logf(a[idx]);
}

void Tensor::logGpu() {
    logKernel<<<(totalSize + 255)/256, 256>>>(gpuData, totalSize);
    hipDeviceSynchronize();
}

__global__ void zeroGradKernel(float* data, int size) {
    int idx = blockIdx.x * blockDim.x + threadIdx.x;
    if (idx < size) {
        data[idx] = 0;
    }
}

void Tensor::zeroGradGpu() {
    int threadsPerBlock = 256;
    int blocks = (totalSize + threadsPerBlock - 1) / threadsPerBlock;
    zeroGradKernel<<<blocks, threadsPerBlock>>>(gpuGrad, totalSize);
    hipDeviceSynchronize();
}

void Tensor::freeGpuMemory() {
    if (gpuData) {
        hipError_t err = hipFree(gpuData);
        if (err != hipSuccess) {
            LOG_CUDA_OP("FREE", "gpuData", 0, 0, false, hipGetErrorString(err));
        } else {
            LOG_MEMORY_DEALLOC("GPU", totalSize * sizeof(float), "Tensor data");
        }
        gpuData = nullptr;
    }

    if (gpuGrad) {
        hipError_t err = hipFree(gpuGrad);
        if (err != hipSuccess) {
            LOG_CUDA_OP("FREE", "gpuGrad", 0, 0, false, hipGetErrorString(err));
        } else {
            LOG_MEMORY_DEALLOC("GPU", totalSize * sizeof(float), "Tensor gradients");
        }
        gpuGrad = nullptr;
    }
}


void Tensor::toGpu() {
    if (gpuData != nullptr) return;
    
    hipError_t err = hipMalloc(&gpuData, totalSize * sizeof(float));
    if (err != hipSuccess) {
        LOG_CUDA_OP("MALLOC", "toGpu", 0, 0, false, hipGetErrorString(err));
        return;
    }
    LOG_MEMORY_ALLOC("GPU", totalSize * sizeof(float), "Tensor data");
    
    err = hipMalloc(&gpuGrad, totalSize * sizeof(float));
    if (err != hipSuccess) {
        LOG_CUDA_OP("MALLOC", "toGpu", 0, 0, false, hipGetErrorString(err));
        hipFree(gpuData);
        return;
    }
    LOG_MEMORY_ALLOC("GPU", totalSize * sizeof(float), "Tensor gradients");
    
    err = hipMemcpy(gpuData, cpuData.data(), totalSize * sizeof(float), hipMemcpyHostToDevice);
    if (err != hipSuccess) {
        LOG_CUDA_OP("MEMCPY", "toGpu", 0, 0, false, hipGetErrorString(err));
        hipFree(gpuData);
        hipFree(gpuGrad);
        return;
    }
    
    LOG_DEBUG("Successfully moved tensor to GPU");
    device = Device::GPU;
}

void Tensor::toCpu() {
    if (gpuData == nullptr) return;
    hipMemcpy(cpuData.data(), gpuData, totalSize * sizeof(float), hipMemcpyDeviceToHost);
    device = Device::CPU;
}

void Tensor::copyCpu() {
    if (gpuData == nullptr) return;
    hipMemcpy(cpuData.data(), gpuData, totalSize * sizeof(float), hipMemcpyDeviceToHost);
}

void Tensor::copyGpu() {
    if (gpuData == nullptr) return;
    hipMemcpy(gpuData, cpuData.data(), totalSize * sizeof(float), hipMemcpyHostToDevice);
}

